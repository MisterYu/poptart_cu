#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void computeGold(char *reference, char *idata, const unsigned int len);
extern "C" void computeGold2(int2 *reference, int2 *idata, const unsigned int len);

///////////////////////////////////////////////////////////////////////////////
//! refract ray with Snell's
//! @param theta1  incident angle of incident ray
//! @param v1 wavefront velocity of incident ray
//! @param v2 wavefront velocity of refracted ray
///////////////////////////////////////////////////////////////////////////////
__device__ float
refract(float theta1, float v1, float v2)
{
    float arg_asin = v2 * sin(theta1) / v1;
    return asin(arg_asin);
}

///////////////////////////////////////////////////////////////////////////////
//! Demonstration of acoustic ray trace
//! @param rays  rays to trace (in and out)
//! @param atms
//! @param n_atms
///////////////////////////////////////////////////////////////////////////////
__global__ void
acoustic_trace_kernel(float3 *rays, float3 *atms, const int n_atms)
{
	// get current ray ID
	const unsigned int tid = threadIdx.x;
	float3 ray = rays[tid];
    if (tid == 5) {
        std::printf("%f %f %f\n", ray.x, ray.y, ray.z);
    }
	for (int i_atm = 0; i_atm < n_atms; i_atm++) {
        float h_layer = atms[i_atm].z;
        float d_layer = h_layer / cos(ray.x);
        // update attentuation
        ray.y -= d_layer * atms[i_atm].y + 20.0 * log10(d_layer);
        // update projected radii/ ground distance
        ray.z += sqrt(d_layer*d_layer - h_layer*h_layer);
        // update angle
        if (i_atm < n_atms - 1)
            ray.x = refract(ray.x, atms[i_atm].x, atms[i_atm + 1].x);
        if (tid == 5) {
            std::printf("%f %f %f\n", ray.x, ray.y, ray.z);
        }
	}
    // write data to global memory
    rays[tid] = ray;
}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool
runAcousticTest(const int argc, const char **argv, 
	float3 *h_rays, unsigned int n_rays, float3 *h_atm_layers, unsigned int n_atm_layers)	
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	findCudaDevice(argc, (const char **)argv);

	const unsigned int num_threads = n_rays;
	const unsigned int rays_mem_size = sizeof(float3) * n_rays;
	const unsigned int atms_mem_size = sizeof(float3) * n_atm_layers;

	// allocate device memory
	float3 *d_rays;
	checkCudaErrors(hipMalloc((void **)&d_rays, rays_mem_size));
	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_rays, h_rays, rays_mem_size,
		hipMemcpyHostToDevice));
	// allocate device memory for int2 version
	float3 *d_atm_layers;
	checkCudaErrors(hipMalloc((void **)&d_atm_layers, atms_mem_size));
	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_atm_layers, h_atm_layers, atms_mem_size,
		hipMemcpyHostToDevice));

	// setup execution parameters
	dim3 grid(1, 1, 1);
	dim3 threads(num_threads, 1, 1);
							  
    // execute the kernel
    acoustic_trace_kernel <<< grid, threads >>>(d_rays, d_atm_layers, n_atm_layers);
	
	// check if kernel execution generated and error
	getLastCudaError("Kernel execution failed");

	// copy results from device to host
	checkCudaErrors(hipMemcpy(h_rays, d_rays, rays_mem_size,
		hipMemcpyDeviceToHost));

	// cleanup memory
	checkCudaErrors(hipFree(d_rays));
	checkCudaErrors(hipFree(d_atm_layers));
	
	return true;
}
